#include "hip/hip_runtime.h"
/**
 * Game of Life Database Generator - d_matrix_2 version
 * Conway's Game of Life 패턴 생성 및 라벨링을 위한 데이터베이스 생성기
 */

#include "GOLdatabase_2.hpp"

namespace GOL_2 {
    using namespace d_matrix_ver2;

    #define MAXGEN 2500
    namespace fs = std::filesystem;

    const std::string DATASET_PATH = "../dataset/";

    const int BOARDWIDTH = 100;
    const int BOARDHEIGHT = 100;
    const int WIDTH = 10;
    const int HEIGHT = 10;

    // CUDA kernel: Game of Life 다음 세대 계산
    __global__ void nextGenKernel(int* current, int* next, int width, int height) {
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < height && j < width) {
            int alive = 0;
            for (int dx = -1; dx <= 1; ++dx) {
                for (int dy = -1; dy <= 1; ++dy) {
                    if(dx == 0 && dy == 0) continue;
                    int ni = i + dx;
                    int nj = j + dy;
                    if (ni >= 0 && ni < height && nj >= 0 && nj < width) {
                        alive += current[ni * width + nj];
                    }
                }
            }

            int idx = i * width + j;
            if (current[idx] == 1) {
                next[idx] = (alive == 2 || alive == 3) ? 1 : 0;
            } else {
                next[idx] = (alive == 3) ? 1 : 0;
            }
        }
    }

    d_matrix_2<int> nextGen(const d_matrix_2<int>& current, hipStream_t str) {
        d_matrix_2<int> next(current.getRow(), current.getCol(), str);
        int* d_curr = current.getDevPointer();
        int* d_next = next.getDevPointer();

        dim3 blockSize(32, 32);
        dim3 gridSize((current.getCol() + 31) / 32, (current.getRow() + 31) / 32);

        nextGenKernel<<<gridSize, blockSize, 0, str>>>(d_curr, d_next, current.getCol(), current.getRow());
        hipStreamSynchronize(str);
        
        return next;
    }

    __global__ void placePatternKernel(int* board, int* pattern, int fullHeight, int fullWidth,
        int patternHeight, int patternWidth,
        int startRow, int startCol) {
        int i = blockIdx.y * blockDim.y + threadIdx.y; // pattern row
        int j = blockIdx.x * blockDim.x + threadIdx.x; // pattern col

        if (i < patternHeight && j < patternWidth) {
            int boardIdx = (startRow + i) * fullWidth + (startCol + j);
            int patternIdx = i * patternWidth + j;
            board[boardIdx] = pattern[patternIdx];
        }
    }

    d_matrix_2<int> generateFixedRatioPatternWithPadding(int fullHeight, int fullWidth, int patternHeight, int patternWidth, double aliveRatio, hipStream_t str) {
        // 1. CPU에서 pattern 배열 셔플
        int totalPatternCells = patternHeight * patternWidth;
        int aliveCells = static_cast<int>(totalPatternCells * aliveRatio);
        std::vector<int> host_pattern(totalPatternCells, 0);
        std::fill_n(host_pattern.begin(), aliveCells, 1);

        std::random_device rd;
        std::mt19937 gen(rd());
        std::shuffle(host_pattern.begin(), host_pattern.end(), gen);

        // 2. GPU 메모리로 복사
        thrust::device_vector<int> d_pattern = host_pattern;
        d_matrix_2<int> board(fullHeight, fullWidth, str); // 전체 보드
        board.fill(0, str); // 0으로 초기화

        int startRow = (fullHeight - patternHeight) / 2;
        int startCol = (fullWidth - patternWidth) / 2;

        // 3. 커널로 중앙에 패턴 복사
        dim3 blockSize(16, 16);
        dim3 gridSize((patternWidth + 15) / 16, (patternHeight + 15) / 16);

        placePatternKernel<<<gridSize, blockSize, 0, str>>>(
            board.getDevPointer(), 
            thrust::raw_pointer_cast(d_pattern.data()), 
            fullHeight, fullWidth, 
            patternHeight, patternWidth, 
            startRow, startCol
        );

        hipStreamSynchronize(str);
        return board;
    }

    __global__ void countAliveKernel(int* mat, int* partialSums, int totalSize) {
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        int localSum = 0;

        for (int i = tid; i < totalSize; i += stride) {
            localSum += mat[i];
        }

        if (tid < totalSize) {
            partialSums[tid] = localSum;
        }
    }

    int countAlive(const d_matrix_2<int>& mat, hipStream_t str) {
        int totalSize = mat.getRow() * mat.getCol();
        int threadsPerBlock = 256;
        int numBlocks = (totalSize + threadsPerBlock - 1) / threadsPerBlock;
        int totalThreads = threadsPerBlock * numBlocks;

        int* d_partialSums;
        hipMallocAsync(&d_partialSums, sizeof(int) * totalThreads, str);

        countAliveKernel<<<numBlocks, threadsPerBlock, 0, str>>>(mat.getDevPointer(), d_partialSums, totalSize);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        }
        hipStreamSynchronize(str);

        std::vector<int> partialSums(totalThreads);
        hipMemcpyAsync(partialSums.data(), d_partialSums, sizeof(int) * totalThreads, hipMemcpyDeviceToHost, str);
        hipStreamSynchronize(str);

        int total = std::accumulate(partialSums.begin(), partialSums.end(), 0);
        hipFreeAsync(d_partialSums, str);

        return total;
    }

    int simulateAndLabel(const d_matrix_2<int>& initialPattern, int fileId, hipStream_t str) {
        d_matrix_2<int> sim = initialPattern;
        std::deque<int> history; // 최근 50개 alive 수 저장
        const int window = 50;

        int constantCount = 0;
        int prev = -1;
        bool strictlyIncreasing = true;
        int gen = 0;

        while (gen < MAXGEN) {
            int alive = countAlive(sim, str);

            // history 갱신
            if (history.size() >= window) history.pop_front();
            history.push_back(alive);

            if (prev == alive) constantCount++;
            else constantCount = 0;

            if (prev != -1 && alive <= prev) strictlyIncreasing = false;
            if (constantCount >= 100 || (strictlyIncreasing && gen >= 100)) break;

            prev = alive;
            sim = nextGen(sim, str);
            gen++;
        }

        return countAlive(sim, str);
    }

    d_matrix_2<int> simulateAndLabelingtopattern(const d_matrix_2<int>& initialPattern, int fileId, hipStream_t str) {
        d_matrix_2<int> sim = initialPattern;
        std::deque<int> history; // 최근 50개 alive 수 저장
        const int window = 50;

        int constantCount = 0;
        int prev = -1;
        bool strictlyIncreasing = true;
        int gen = 0;

        while (gen < MAXGEN) {
            int alive = countAlive(sim, str);

            // history 갱신
            if (history.size() >= window) history.pop_front();
            history.push_back(alive);

            if (prev == alive) constantCount++;
            else constantCount = 0;

            if (prev != -1 && alive <= prev) strictlyIncreasing = false;
            if (constantCount >= 100 || (strictlyIncreasing && gen >= 100)) break;

            prev = alive;
            sim = nextGen(sim, str);
            gen++;
        }

        return sim;
    }

    void generateGameOfLifeData(int filenum, double ratio) {
        int deviceCount = 0;
        hipError_t err = hipGetDeviceCount(&deviceCount);
        if (err != hipSuccess || deviceCount == 0) {
            std::cerr << "[FATAL] No CUDA device: " << hipGetErrorString(err) << std::endl;
            exit(1);
        }
        hipSetDevice(0);

        // 스트림 생성
        hipStream_t stream;
        hipStreamCreate(&stream);

        fs::create_directories(DATASET_PATH);

        int totalFiles = filenum;
        double aliveratio = ratio;

        std::cout << "totalFiles:" << totalFiles << " (file direction: ../dataset)" << std::endl;
        std::cout << "aliveratio:" << aliveratio << std::endl;
        std::cout << "max generation:" << MAXGEN << std::endl;
        std::cout << "pattern size:" << HEIGHT << " * " << WIDTH << std::endl;
        std::cout << "board size:" << BOARDHEIGHT << " * " << BOARDWIDTH << std::endl;

        const char *command1 = "find ../dataset/ -type f -delete";
        std::system(command1);

        auto startTime = std::chrono::steady_clock::now();

        for (int fileId = 1; fileId <= totalFiles; ++fileId) {
            int label = -1;
            d_matrix_2<int> pattern = generateFixedRatioPatternWithPadding(BOARDHEIGHT, BOARDWIDTH, HEIGHT, WIDTH, aliveratio, stream);
            d_matrix_2<int> last_pattern = simulateAndLabelingtopattern(pattern, fileId, stream);
            label = simulateAndLabel(pattern, fileId, stream);

            std::ofstream fout(DATASET_PATH + "sample" + std::to_string(fileId) + ".txt");

            int startRow = (BOARDHEIGHT - HEIGHT) / 2;
            int startCol = (BOARDWIDTH - WIDTH) / 2;

            // 초기 패턴을 호스트로 복사
            pattern.cpyToHost();

            // 초기 패턴 저장
            for (int i = startRow; i < startRow + HEIGHT; ++i) {
                for (int j = startCol; j < startCol + WIDTH; ++j) {
                    fout << pattern(i, j);
                }
                fout << '\n';
            }

            fout << label << '\n';
            fout << '\n';

            // 최종 패턴을 호스트로 복사
            last_pattern.cpyToHost();

            // 최종 패턴 저장
            for(int i = 0; i < BOARDHEIGHT; i++){
                for(int j = 0; j < BOARDWIDTH; j++){
                    fout << last_pattern(i, j);
                }
                fout << '\n';
            }

            fout.close();
            printProgressBar(fileId, totalFiles, startTime, "");
        }
        
        std::cout << std::endl << "[Done] Dataset generation complete." << std::endl;

        auto totalElapsed = std::chrono::steady_clock::now() - startTime;
        int totalSec = std::chrono::duration_cast<std::chrono::seconds>(totalElapsed).count();
        std::cout << "총 실행 시간: " << totalSec << " 초" << std::endl;

        hipStreamDestroy(stream);
    }

    std::vector<std::pair<d_matrix_2<double>, d_matrix_2<double>>> LoadingData() {
        std::vector<std::pair<d_matrix_2<double>, d_matrix_2<double>>> dataset;
        dataset.reserve(1000);

        // 스트림 생성
        hipStream_t stream;
        hipStreamCreate(&stream);

        for (const auto& entry : fs::directory_iterator(DATASET_PATH)) {
            if (entry.path().extension() != ".txt") continue;

            std::ifstream fin(entry.path());
            if (!fin) {
                std::cerr << "파일 열기 실패: " << entry.path() << '\n';
                continue;
            }

            d_matrix_2<double> input(WIDTH*HEIGHT, 1, stream);
            std::string line;
            int row = 0;
            while (row < WIDTH && std::getline(fin, line)) {
                int len = std::min(HEIGHT, static_cast<int>(line.size()));
                for (int col = 0; col < len; ++col) {
                    input(row * HEIGHT + col, 0) = line[col] - '0';
                }
                row++;
            }

            int label_index = -1;
            if (std::getline(fin, line)) label_index = std::stoi(line);

            d_matrix_2<double> label(BIT_WIDTH, 1, stream);
            // 1) 모두 0으로 초기화
            label.fill(0.0, stream);
            // 2) 각 비트 위치에 0/1 설정 (LSB부터)
            for (int b = 0; b < BIT_WIDTH; ++b) {
                label(b, 0) = (label_index >> b) & 1;
            }

            input.cpyToDev();
            label.cpyToDev();
            dataset.emplace_back(std::move(input), std::move(label));
        }

        hipStreamDestroy(stream);
        return dataset;
    }

} // namespace GOL_2
