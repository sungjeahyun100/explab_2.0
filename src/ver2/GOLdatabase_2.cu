#include "hip/hip_runtime.h"
/**
 * Game of Life Database Generator - d_matrix_2 version
 * Conway's Game of Life 패턴 생성 및 라벨링을 위한 데이터베이스 생성기
 */

#include "GOLdatabase_2.hpp"

namespace GOL_2 {
    using namespace d_matrix_ver2;

    #define MAXGEN 2500
    namespace fs = std::filesystem;

    const std::string DATASET_PATH = "../dataset/";

    const int BOARDWIDTH = 100;
    const int BOARDHEIGHT = 100;
    const int WIDTH = 10;
    const int HEIGHT = 10;

    // CUDA kernel: Game of Life 다음 세대 계산
    __global__ void nextGenKernel(int* current, int* next, int width, int height) {
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < height && j < width) {
            int alive = 0;
            for (int dx = -1; dx <= 1; ++dx) {
                for (int dy = -1; dy <= 1; ++dy) {
                    if(dx == 0 && dy == 0) continue;
                    int ni = i + dx;
                    int nj = j + dy;
                    if (ni >= 0 && ni < height && nj >= 0 && nj < width) {
                        alive += current[ni * width + nj];
                    }
                }
            }

            int idx = i * width + j;
            if (current[idx] == 1) {
                next[idx] = (alive == 2 || alive == 3) ? 1 : 0;
            } else {
                next[idx] = (alive == 3) ? 1 : 0;
            }
        }
    }

    d_matrix_2<int> nextGen(const d_matrix_2<int>& current, hipStream_t str) {
        d_matrix_2<int> next(current.getRow(), current.getCol(), str);
        int* d_curr = current.getDevPointer();
        int* d_next = next.getDevPointer();

        dim3 blockSize(32, 32);
        dim3 gridSize((current.getCol() + 31) / 32, (current.getRow() + 31) / 32);

        nextGenKernel<<<gridSize, blockSize, 0, str>>>(d_curr, d_next, current.getCol(), current.getRow());
        hipStreamSynchronize(str);
        
        return next;
    }

    __global__ void placePatternKernel(int* board, int* pattern, int fullHeight, int fullWidth,
        int patternHeight, int patternWidth,
        int startRow, int startCol) {
        int i = blockIdx.y * blockDim.y + threadIdx.y; // pattern row
        int j = blockIdx.x * blockDim.x + threadIdx.x; // pattern col

        if (i < patternHeight && j < patternWidth) {
            int boardIdx = (startRow + i) * fullWidth + (startCol + j);
            int patternIdx = i * patternWidth + j;
            board[boardIdx] = pattern[patternIdx];
        }
    }

    d_matrix_2<int> generateFixedRatioPatternWithPadding(int fullHeight, int fullWidth, int patternHeight, int patternWidth, double aliveRatio, hipStream_t str) {
        // 1. CPU에서 pattern 배열 셔플
        int totalPatternCells = patternHeight * patternWidth;
        int aliveCells = static_cast<int>(totalPatternCells * aliveRatio);
        std::vector<int> host_pattern(totalPatternCells, 0);
        std::fill_n(host_pattern.begin(), aliveCells, 1);

        std::random_device rd;
        std::mt19937 gen(rd());
        std::shuffle(host_pattern.begin(), host_pattern.end(), gen);

        // 2. GPU 메모리로 복사
        thrust::device_vector<int> d_pattern = host_pattern;
        d_matrix_2<int> board(fullHeight, fullWidth, str); // 전체 보드
        board.fill(0, str); // 0으로 초기화

        int startRow = (fullHeight - patternHeight) / 2;
        int startCol = (fullWidth - patternWidth) / 2;

        // 3. 커널로 중앙에 패턴 복사
        dim3 blockSize(16, 16);
        dim3 gridSize((patternWidth + 15) / 16, (patternHeight + 15) / 16);

        placePatternKernel<<<gridSize, blockSize, 0, str>>>(
            board.getDevPointer(), 
            thrust::raw_pointer_cast(d_pattern.data()), 
            fullHeight, fullWidth, 
            patternHeight, patternWidth, 
            startRow, startCol
        );

        hipStreamSynchronize(str);
        return board;
    }

    __global__ void countAliveKernel(int* mat, int* partialSums, int totalSize) {
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        int localSum = 0;

        for (int i = tid; i < totalSize; i += stride) {
            localSum += mat[i];
        }

        if (tid < totalSize) {
            partialSums[tid] = localSum;
        }
    }

    int countAlive(const d_matrix_2<int>& mat, hipStream_t str) {
        int totalSize = mat.getRow() * mat.getCol();
        int threadsPerBlock = 256;
        int numBlocks = (totalSize + threadsPerBlock - 1) / threadsPerBlock;
        int totalThreads = threadsPerBlock * numBlocks;

        int* d_partialSums;
        hipMallocAsync(&d_partialSums, sizeof(int) * totalThreads, str);

        countAliveKernel<<<numBlocks, threadsPerBlock, 0, str>>>(mat.getDevPointer(), d_partialSums, totalSize);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        }
        hipStreamSynchronize(str);

        std::vector<int> partialSums(totalThreads);
        hipMemcpyAsync(partialSums.data(), d_partialSums, sizeof(int) * totalThreads, hipMemcpyDeviceToHost, str);
        hipStreamSynchronize(str);

        int total = std::accumulate(partialSums.begin(), partialSums.end(), 0);
        hipFreeAsync(d_partialSums, str);

        return total;
    }

    int simulateAndLabel(const d_matrix_2<int>& initialPattern, int fileId, hipStream_t str) {
        d_matrix_2<int> sim = initialPattern;
        std::deque<int> history; // 최근 50개 alive 수 저장
        const int window = 50;

        int constantCount = 0;
        int prev = -1;
        bool strictlyIncreasing = true;
        int gen = 0;

        while (gen < MAXGEN) {
            int alive = countAlive(sim, str);

            // history 갱신
            if (history.size() >= window) history.pop_front();
            history.push_back(alive);

            if (prev == alive) constantCount++;
            else constantCount = 0;

            if (prev != -1 && alive <= prev) strictlyIncreasing = false;
            if (constantCount >= 100 || (strictlyIncreasing && gen >= 100)) break;

            prev = alive;
            sim = nextGen(sim, str);
            gen++;
        }

        return countAlive(sim, str);
    }

    d_matrix_2<int> simulateAndLabelingtopattern(const d_matrix_2<int>& initialPattern, int fileId, hipStream_t str) {
        d_matrix_2<int> sim = initialPattern;
        std::deque<int> history; // 최근 50개 alive 수 저장
        const int window = 50;

        int constantCount = 0;
        int prev = -1;
        bool strictlyIncreasing = true;
        int gen = 0;

        while (gen < MAXGEN) {
            int alive = countAlive(sim, str);

            // history 갱신
            if (history.size() >= window) history.pop_front();
            history.push_back(alive);

            if (prev == alive) constantCount++;
            else constantCount = 0;

            if (prev != -1 && alive <= prev) strictlyIncreasing = false;
            if (constantCount >= 100 || (strictlyIncreasing && gen >= 100)) break;

            prev = alive;
            sim = nextGen(sim, str);
            gen++;
        }

        return sim;
    }

    // 최적화: 단일 시뮬레이션으로 패턴과 라벨을 동시에 반환
    std::pair<d_matrix_2<int>, int> simulateAndGetBoth(const d_matrix_2<int>& initialPattern, int fileId, hipStream_t str) {
        d_matrix_2<int> sim = initialPattern;
        std::deque<int> history; // 최근 50개 alive 수 저장
        const int window = 50;

        int constantCount = 0;
        int prev = -1;
        bool strictlyIncreasing = true;
        int gen = 0;

        while (gen < MAXGEN) {
            int alive = countAlive(sim, str);

            // history 갱신
            if (history.size() >= window) history.pop_front();
            history.push_back(alive);

            if (prev == alive) constantCount++;
            else constantCount = 0;

            if (prev != -1 && alive <= prev) strictlyIncreasing = false;
            
            // 더 빠른 조기 종료: 안정화 감지 개선
            if (constantCount >= 30) break;  // 30 세대 연속 동일 → 안정화
            if (strictlyIncreasing && gen >= 50) break;  // 50 세대 연속 증가 → 발산
            if (alive == 0) break;  // 모든 셀 사망 → 소멸
            
            // 진동 패턴 감지: 최근 기록에서 반복 확인
            if (history.size() >= 20) {
                bool oscillating = true;
                int period = 2;  // 2주기 진동 확인
                for (int i = 0; i < 10 && oscillating; i++) {
                    if (history[history.size()-1-i] != history[history.size()-1-i-period]) {
                        oscillating = false;
                    }
                }
                if (oscillating) break;  // 진동 패턴 감지 시 조기 종료
            }

            prev = alive;
            sim = nextGen(sim, str);
            gen++;
        }

        int final_count = countAlive(sim, str);
        return {std::move(sim), final_count};
    }

    void generateGameOfLifeData(int filenum, double ratio) {
        int deviceCount = 0;
        hipError_t err = hipGetDeviceCount(&deviceCount);
        if (err != hipSuccess || deviceCount == 0) {
            std::cerr << "[FATAL] No CUDA device: " << hipGetErrorString(err) << std::endl;
            exit(1);
        }
        hipSetDevice(0);

        // 스트림 생성
        hipStream_t stream;
        hipStreamCreate(&stream);

        fs::create_directories(DATASET_PATH);

        int totalFiles = filenum;
        double aliveratio = ratio;

        std::cout << "totalFiles:" << totalFiles << " (file direction: " << DATASET_PATH << ")" << std::endl;
        std::cout << "aliveratio:" << aliveratio << std::endl;
        std::cout << "max generation:" << MAXGEN << std::endl;
        std::cout << "pattern size:" << HEIGHT << " * " << WIDTH << std::endl;
        std::cout << "board size:" << BOARDHEIGHT << " * " << BOARDWIDTH << std::endl;

        const char *command1 = "find ../dataset/ -type f -delete";
        std::system(command1);

        auto startTime = std::chrono::steady_clock::now();

        for (int fileId = 1; fileId <= totalFiles; ++fileId) {
            d_matrix_2<int> pattern = generateFixedRatioPatternWithPadding(BOARDHEIGHT, BOARDWIDTH, HEIGHT, WIDTH, aliveratio, stream);
            
            // 단일 시뮬레이션으로 최종 패턴과 라벨을 동시에 얻음
            auto [last_pattern, label] = simulateAndGetBoth(pattern, fileId, stream);

            std::ofstream fout(DATASET_PATH + "sample" + std::to_string(fileId) + ".txt");

            int startRow = (BOARDHEIGHT - HEIGHT) / 2;
            int startCol = (BOARDWIDTH - WIDTH) / 2;

            // 초기 패턴을 호스트로 복사 (비동기)
            pattern.cpyToHost(stream);
            
            // GPU 작업이 완료될 때까지 대기
            hipStreamSynchronize(stream);

            // 초기 패턴 저장
            for (int i = startRow; i < startRow + HEIGHT; ++i) {
                for (int j = startCol; j < startCol + WIDTH; ++j) {
                    fout << pattern(i, j);
                }
                fout << '\n';
            }

            fout << label << '\n';
            fout << '\n';

            // 최종 패턴을 호스트로 복사 (비동기)
            last_pattern.cpyToHost(stream);
            hipStreamSynchronize(stream);

            // 최종 패턴 저장
            for(int i = 0; i < BOARDHEIGHT; i++){
                for(int j = 0; j < BOARDWIDTH; j++){
                    fout << last_pattern(i, j);
                }
                fout << '\n';
            }

            fout.close();
            
            // 진행률 표시 최적화: 매 10번째마다만 업데이트
            if (fileId % 10 == 0 || fileId == totalFiles) {
                std::string prograss_name = "GOL data generating... " + std::to_string(fileId) + "/" + std::to_string(totalFiles);
                printProgressBar(fileId, totalFiles, startTime, prograss_name);
            }
        }
        
        std::cout << std::endl << "[Done] Dataset generation complete." << std::endl;

        auto totalElapsed = std::chrono::steady_clock::now() - startTime;
        int totalSec = std::chrono::duration_cast<std::chrono::seconds>(totalElapsed).count();
        std::cout << "총 실행 시간: " << totalSec << " 초" << std::endl;

        hipStreamDestroy(stream);
    }

    std::pair<d_matrix_2<double>, d_matrix_2<double>> LoadingDataBatch(hipStream_t str) {
        std::vector<std::pair<d_matrix_2<double>, d_matrix_2<double>>> temp_dataset;
        temp_dataset.reserve(5000);  // 4000개 + 여유분

        // 파일들을 읽어서 임시 벡터에 저장
        for (const auto& entry : fs::directory_iterator(DATASET_PATH)) {
            if (entry.path().extension() != ".txt") continue;

            std::ifstream fin(entry.path());
            if (!fin) {
                std::cerr << "파일 열기 실패: " << entry.path() << '\n';
                continue;
            }

            d_matrix_2<double> input(WIDTH*HEIGHT, 1, str);
            std::string line;
            int row = 0;
            while (row < WIDTH && std::getline(fin, line)) {
                int len = std::min(HEIGHT, static_cast<int>(line.size()));
                for (int col = 0; col < len; ++col) {
                    input(row * HEIGHT + col, 0) = line[col] - '0';
                }
                row++;
            }

            int label_index = -1;
            if (std::getline(fin, line)) label_index = std::stoi(line);

            d_matrix_2<double> label(BIT_WIDTH, 1, str);
            // 1) 모두 0으로 초기화
            label.fill(0.0, str);
            // 2) 각 비트 위치에 0/1 설정 (LSB부터)
            for (int b = 0; b < BIT_WIDTH; ++b) {
                label(b, 0) = (label_index >> b) & 1;
            }

            input.cpyToDev(str);
            label.cpyToDev(str);
            temp_dataset.emplace_back(std::move(input), std::move(label));
        }

        int N = temp_dataset.size();
        std::cout << "로드된 데이터 개수: " << N << std::endl;

        // 배치 친화적인 형태로 변환: (N, features) 형태의 행렬 2개
        d_matrix_2<double> X(N, WIDTH*HEIGHT, str);  // 입력 행렬: (샘플수, 특성수)
        d_matrix_2<double> Y(N, BIT_WIDTH, str);     // 라벨 행렬: (샘플수, 클래스수)

        // 데이터 복사
        for (int i = 0; i < N; i++) {
            temp_dataset[i].first.cpyToHost(str);
            temp_dataset[i].second.cpyToHost(str);
            hipStreamSynchronize(str);

            // 입력 데이터 복사
            for (int j = 0; j < WIDTH*HEIGHT; j++) {
                X(i, j) = temp_dataset[i].first.getHostPointer()[j];
            }

            // 라벨 데이터 복사
            for (int j = 0; j < BIT_WIDTH; j++) {
                Y(i, j) = temp_dataset[i].second.getHostPointer()[j];
            }
        }

        // GPU로 전송
        X.cpyToDev(str);
        Y.cpyToDev(str);
        hipStreamSynchronize(str);

        return {std::move(X), std::move(Y)};
    }

} // namespace GOL_2
