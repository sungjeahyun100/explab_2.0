#include "hip/hip_runtime.h"
#include "perceptron.hpp"

// 현재 시간 문자열 반환 (가중치 저장 파일명에 사용)
std::string getCurrentTimestamp()
{
    auto now = std::chrono::system_clock::now();
    std::time_t t_now = std::chrono::system_clock::to_time_t(now);
    std::tm tm_now = *std::localtime(&t_now);

    std::ostringstream oss;
    oss << std::put_time(&tm_now, "%Y-%m-%d_%H%M%S");
    return oss.str();
}

// convolutionLayer 구현
convolutionLayer::convolutionLayer(int iRow, int iCol, int fRow, int fCol,
                                   int nFilter, double lr, InitType init)
    : inRow(iRow), inCol(iCol), kRow(fRow), kCol(fCol), numFilter(nFilter),
      outRow(iRow - fRow + 1), outCol(iCol - fCol + 1), learning_rate(lr),
      input(iRow, iCol), bias(nFilter, 1),
      flatOutput(nFilter * (iRow - fRow + 1) * (iCol - fCol + 1), 1)
{
    bias.fill(0.0);
    kernels.reserve(numFilter);
    outputs.reserve(numFilter);
    for(int i=0;i<numFilter;i++){
        kernels.emplace_back(kRow, kCol);
        kernels[i] = InitWeight<double>(kRow, kCol, init);
        outputs.emplace_back(outRow, outCol);
    }
}

void convolutionLayer::feedforward(const d_matrix<double>& raw_input)
{
    input = raw_input;
    for(int f=0; f<numFilter; ++f){
        outputs[f] = convolute(input, kernels[f]);
        outputs[f] = ScalaPlus(outputs[f], bias(f,0));
    }

    int idx = 0;
    for(int f=0; f<numFilter; ++f){
        for(int r=0; r<outRow; ++r){
            for(int c=0; c<outCol; ++c){
                flatOutput(idx++,0) = outputs[f](r,c);
            }
        }
    }
    flatOutput.cpyToDev();
}

void convolutionLayer::backprop(const d_matrix<double>& delta_flat)
{
    int idx = 0;
    for(int f=0; f<numFilter; ++f){
        d_matrix<double> d_out(outRow, outCol);
        for(int r=0; r<outRow; ++r){
            for(int c=0; c<outCol; ++c){
                d_out(r,c) = delta_flat(idx++,0);
            }
        }

        d_matrix<double> grad_k = convolute(input, d_out);
        kernels[f] = matrixPlus(kernels[f], ScalaProduct(grad_k, -learning_rate));
        double b_grad = plusAllElements(d_out);
        bias(f,0) -= learning_rate * b_grad;
    }
}

d_matrix<double>& convolutionLayer::getOutput()
{
    return flatOutput;
}

// 가중치 파일에서 weight, bias 불러오기
void perceptronLayer::loadWeight(const std::string &path)
{
    std::ifstream test_subject(path, std::ios::binary);
    if (!test_subject) {
        std::cerr << "Error opening file: " << path << std::endl;
        return;
    }

    test_subject >> weight;
    test_subject >> bias;

    test_subject.close();
}

// 가중치 파일로 저장 (이름: subject+타임스탬프)
void perceptronLayer::saveWeight() {
    // 1) 파일명: subject_YYYYMMDD_HHMMSS.bin
    std::string ts = getCurrentTimestamp();
    std::string path = WEIGHT_DATAPATH + "subject_" + ts + ".bin";

    // 2) 바이너리 모드로 열기
    std::ofstream ofs(path, std::ios::binary);
    if (!ofs) {
        std::cerr << "가중치 파일 열기 실패: " << path << "\n";
        return;
    }

    // 3) weight, bias 덤프
    //    assuming weight and bias are contiguous (e.g., std::vector<double> or d_matrix)
    size_t wCount = weight.size();      // 전체 요소 수
    size_t bCount = bias.size();

    // 먼저 요소 개수 기록 (나중에 로드할 때 도움이 됩니다)
    ofs.write(reinterpret_cast<const char*>(&wCount), sizeof(wCount));
    ofs.write(reinterpret_cast<const char*>(&bCount), sizeof(bCount));

    // 실제 데이터 기록
    ofs.write(reinterpret_cast<const char*>(weight.getHostPointer()), wCount * sizeof(double));
    ofs.write(reinterpret_cast<const char*>(bias.getHostPointer()),  bCount * sizeof(double));

    ofs.close();
    std::cout << "  → 가중치 저장 완료: " << path << "\n";
}

d_matrix<double>& perceptronLayer::getOutput() { return output; }

// weight, bias를 GPU로 복사
void perceptronLayer::updateWeightInDev() {
    weight.cpyToDev();
    bias.cpyToDev();
}

// feedforward: z = W x + b, output = z
// (활성화는 ActivateLayer에서 적용)
void perceptronLayer::feedforward(const d_matrix<double>& raw_input) {
    input = raw_input;

    z = matrixPlus(matrixMP(weight, input), bias);

    output = z;
}

// 그래디언트 계산 (델타, Gt_W, Gt_B)
// 델타: δ = (next->weight^T * next->delta) ⊙ act_deriv
// Gt_W = δ * input^T, Gt_B = δ
void perceptronLayer::calculateGrad(perceptronLayer* next, const d_matrix<double>& external_delta, const d_matrix<double>& act_deriv) {
    d_matrix<double> grad_input = external_delta;

    if (next != nullptr) {
        d_matrix<double> weighted_delta = matrixMP(next->weight.transpose(), next->delta);
        weighted_delta.cpyToDev();
        grad_input = weighted_delta;
    }

    delta = HadamardProduct(grad_input, act_deriv);
    delta.cpyToDev();

    Gt_W = matrixMP<double>(delta, input.transpose());
    Gt_B = delta;

    hipDeviceSynchronize();
}

// 입력 설정 (input = in)
void ActivateLayer::pushInput(const d_matrix<double>& in){
    input = in;
    input.cpyToDev();
}

// 활성화 적용 (output = f(input))
// 지원: ReLU, LReLU, Identity, Sigmoid
void ActivateLayer::Active(){
    switch (act) {
        case ActivationType::ReLU:
            output = MatrixActivate<double, relu>(input); break;
        case ActivationType::LReLU:
            output = MatrixActivate<double, lrelu>(input); break;
        case ActivationType::Identity:
            output = MatrixActivate<double, Identity>(input); break;
        case ActivationType::Sigmoid:
            output = MatrixActivate<double, sigmoid>(input); break;
        case ActivationType::Tanh:
            output = MatrixActivate<double, Tanh>(input); break;
        case ActivationType::ELU:
            output = MatrixActivate<double, ELU>(input); break;
        case ActivationType::SELU:
            output = MatrixActivate<double, SELU>(input); break;
        case ActivationType::Softplus:
            output = MatrixActivate<double, Softplus>(input); break;
        case ActivationType::Softsign:
            output = MatrixActivate<double, Softsign>(input); break;
        case ActivationType::Swish:
            output = MatrixActivate<double, Swish>(input); break;
        default:
            throw std::runtime_error("Unsupported ActivationType in perceptronLayer");
    }
}

// 활성화 함수 미분값 반환 (f'(z))
// ReLU: 1(x>0), 0(x<=0)
// LReLU: 1(x>0), 0.01(x<=0)
// Identity: 1
// Sigmoid: σ'(x) = σ(x)(1-σ(x))
d_matrix<double> ActivateLayer::d_Active(const d_matrix<double>& z) {
    switch (act) {
        case ActivationType::ReLU:
            return MatrixActivate<double, d_relu>(z); 
        case ActivationType::LReLU:
            return MatrixActivate<double, d_lrelu>(z); 
        case ActivationType::Identity:
            return MatrixActivate<double, d_I>(z); 
        case ActivationType::Sigmoid:
            return MatrixActivate<double, d_sigmoid>(z); 
        case ActivationType::Tanh:
            return MatrixActivate<double, d_tanh>(z); 
        case ActivationType::ELU:
            return MatrixActivate<double, d_ELU>(z); 
        case ActivationType::SELU:
            return MatrixActivate<double, d_SELU>(z); 
        case ActivationType::Softplus:
            return MatrixActivate<double, sigmoid>(z);
        case ActivationType::Softsign:
            return MatrixActivate<double, d_Softsign>(z);
        case ActivationType::Swish:
            return MatrixActivate<double, d_Swish>(z);
        default:
            throw std::runtime_error("Unsupported ActivationType in d_Active");
    }
}

// 활성화 결과 반환
const d_matrix<double>& ActivateLayer::getOutput() const {
    return output; 
}

// 타겟 입력
void LossLayer::pushTarget(const d_matrix<double>& Target){
    target = Target;
}

// 출력 입력
void LossLayer::pushOutput(const d_matrix<double>& Output){
    output = Output;
}

// 손실값 반환
// MSE: L = 1/n Σ(y-p)^2
// CrossEntropy: L = -Σ y log(softmax(p))
double LossLayer::getLoss(){
    // 1) 디바이스→호스트 복사
    output.cpyToHost();
    target.cpyToHost();

    switch (Loss)
    {
        case LossType::MSE: {
            // MSE: L = 1/N Σ (output − target)², 전부 호스트 계산
            int N = output.getRow();
            double sum = 0.0;
            for (int i = 0; i < N; ++i) {
                double diff = output(i, 0) - target(i, 0);
                sum += diff * diff;
            }
            return sum / static_cast<double>(N);
        }

        case LossType::CrossEntropy: {
            int N = output.getRow();
            // 2) 소프트맥스 확률 계산
            d_matrix<double> p = softmax(output);

            // 3) 크로스엔트로피 손실: L = -1/N Σ y_i * log(p_i)
            double loss = 0.0;
            for (int i = 0; i < N; ++i) {
                double yi = target(i, 0);
                double pi = std::min(std::max(p(i, 0), 1e-12), 1.0);  // 클리핑
                loss -= yi * std::log(pi);
            }
            return loss / static_cast<double>(N);
        }

        default:
            throw std::runtime_error("Unsupported LossType in getLoss");
    }
}

// 손실 미분 반환
// MSE: dL/dz = 2(y-p)
// CrossEntropy: dL/dz = softmax(p) - y
d_matrix<double> LossLayer::getGrad() {
    // 1) 디바이스→호스트 복사
    output.cpyToHost();
    target.cpyToHost();

    switch (Loss) {
        case LossType::MSE: {
            // L = (1/N) Σ (o - t)^2  이므로  dL/dz = 2*(o - t)/N
            int N = output.getRow();
            // diff = output - target
            d_matrix<double> diff = matrixPlus(output, ScalaProduct(target, -1.0));
            return ScalaProduct(diff, 2.0 / static_cast<double>(N));
        }

        case LossType::CrossEntropy: {
            int N = output.getRow();
            // 2) 소프트맥스 확률 계산
            d_matrix<double> p = softmax(output);

            // 3) gradient = (p - y) / N
            d_matrix<double> grad = matrixPlus(p, ScalaProduct(target, -1.0));
            return ScalaProduct(grad, 1.0 / static_cast<double>(N));
        }

        default:
            throw std::runtime_error("Unsupported LossType in getGrad");
    }
}
Adam::~Adam(){}

// Adam 옵티마이저 역전파
// m, v: 1차/2차 모멘트, 베타1/2, epsilon, t(스텝)
// 업데이트 수식:
// m = β₁ m + (1-β₁)g, v = β₂ v + (1-β₂)g²
// m̂ = m/(1-β₁ᵗ), v̂ = v/(1-β₂ᵗ)
// W -= lr * m̂/(sqrt(v̂)+ε)
void Adam::backprop(perceptronLayer* next, const d_matrix<double>& external_delta, const d_matrix<double>& act_deriv){

    this->t++;

    // 1) gradient 계산
    this->calculateGrad(next, external_delta, act_deriv);

    // 2) 1차 및 2차 모멘트 갱신
    this->m_W = matrixPlus(ScalaProduct(this->m_W, this->beta1), ScalaProduct(this->Gt_W, 1.0 - this->beta1));
    this->v_W = matrixPlus(ScalaProduct(this->v_W, this->beta2), ScalaProduct(HadamardProduct(this->Gt_W, this->Gt_W), 1.0 - this->beta2));
    this->m_B = matrixPlus(ScalaProduct(this->m_B, this->beta1), ScalaProduct(this->Gt_B, 1.0 - this->beta1));
    this->v_B = matrixPlus(ScalaProduct(this->v_B, this->beta2), ScalaProduct(HadamardProduct(this->Gt_B, this->Gt_B), 1.0 - this->beta2));

    // 3) 편향 보정 계수
    double bias_corr1 = 1.0 - std::pow(this->beta1, this->t);
    double bias_corr2 = 1.0 - std::pow(this->beta2, this->t);

    // 4) 편향 보정된 모멘트
    d_matrix<double> m_W_hat = ScalaProduct(this->m_W, 1.0 / bias_corr1);
    d_matrix<double> v_W_hat = ScalaProduct(this->v_W, 1.0 / bias_corr2);
    d_matrix<double> m_B_hat = ScalaProduct(this->m_B, 1.0 / bias_corr1);
    d_matrix<double> v_B_hat = ScalaProduct(this->v_B, 1.0 / bias_corr2);

    // 5) 분모: sqrt(v̂) + ε
    //    MatrixActivate<sqrt> 는 elementwise sqrt, devide 는 reciprocal
    auto sqrt_vW = MatrixActivate<double, sqr>(v_W_hat);
    auto denomW  = ScalaPlus(sqrt_vW, this->epsilon);
    auto invDenW = MatrixActivate<double, devide>(denomW);

    auto sqrt_vB = MatrixActivate<double, sqr>(v_B_hat);
    auto denomB  = ScalaPlus(sqrt_vB, this->epsilon);
    auto invDenB = MatrixActivate<double, devide>(denomB);

    // 6) 파라미터 업데이트
    //    w ← w − lr * (m̂ ⊙ invDen)
    this->weight = matrixPlus(
        this->weight,
        ScalaProduct(HadamardProduct(m_W_hat, invDenW), -this->learning_rate)
    );
    this->bias = matrixPlus(
        this->bias,
        ScalaProduct(HadamardProduct(m_B_hat, invDenB), -this->learning_rate)
    );

    // 7) 디바이스 메모리에 복사
    this->updateWeightInDev();
    hipDeviceSynchronize();

}

SGD::~SGD(){}

// SGD 옵티마이저 역전파
// W -= lr * grad
void SGD::backprop(perceptronLayer* next, const d_matrix<double>& external_delta, const d_matrix<double>& act_deriv)
{
    this->calculateGrad(next, external_delta, act_deriv);
    this->weight = matrixPlus(this->weight, ScalaProduct(this->Gt_W, (-1) * this->learning_rate));
    this->bias = matrixPlus(this->bias, ScalaProduct(this->Gt_B, (-1) * this->learning_rate));
    this->updateWeightInDev();
    hipDeviceSynchronize();
}

/*
[MLP(다층 퍼셉트론) 구성 예시]

// 1. 계층 선언 (입력, 은닉, 출력)
SGD input_layer(입크기, 은닉크기, lr, InitType::He);
ActivateLayer act1(은닉크기, 1, ActivationType::ReLU);
SGD output_layer(은닉크기, 출력크기, lr, InitType::He);
ActivateLayer act2(출력크기, 1, ActivationType::Sigmoid); // 또는 Softmax
LossLayer loss(출력크기, 1, LossType::CrossEntropy);

// 2. 순전파 예시
input_layer.feedforward(input); // 첫 계층
act1.pushInput(input_layer.getOutput());
act1.Active();
output_layer.feedforward(act1.getOutput());
act2.pushInput(output_layer.getOutput());
act2.Active();

// 3. 역전파 예시
loss.pushTarget(target);
loss.pushOutput(act2.getOutput());
d_matrix<double> grad = loss.getGrad();
output_layer.backprop(nullptr, grad, act2.d_Active(output_layer.getOutput()));
input_layer.backprop(&output_layer, output_layer.delta, act1.d_Active(input_layer.getOutput()));

[MLP(다층 퍼셉트론) 구성 예시 - Adam 사용]

// 1. 계층 선언 (입력, 은닉, 출력)
Adam input_layer(입크기, 은닉크기, lr, InitType::He);
ActivateLayer act1(은닉크기, 1, ActivationType::ReLU);
Adam output_layer(은닉크기, 출력크기, lr, InitType::He);
ActivateLayer act2(출력크기, 1, ActivationType::Sigmoid); // 또는 Softmax
LossLayer loss(출력크기, 1, LossType::CrossEntropy);

// 2. 순전파 예시
input_layer.feedforward(input); // 첫 계층
act1.pushInput(input_layer.getOutput());
act1.Active();
output_layer.feedforward(act1.getOutput());
act2.pushInput(output_layer.getOutput());
act2.Active();

// 3. 역전파 예시
loss.pushTarget(target);
loss.pushOutput(act2.getOutput());
d_matrix<double> grad = loss.getGrad();
output_layer.backprop(nullptr, grad, act2.d_Active(output_layer.getOutput()));
input_layer.backprop(&output_layer, output_layer.delta, act1.d_Active(input_layer.getOutput()));
*/



