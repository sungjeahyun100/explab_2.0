#include "hip/hip_runtime.h"
#include <ver2/perceptron_2.hpp>
#include <ver2/utility.hpp>
#include <cmath>

namespace p2 = perceptron_2;

/*
    python 
import gdb
m = gdb.parse_and_eval("d_A")                               
rows = int(m['row'])                                      
cols = int(m['col'])                                      
for i in range(rows):                                     
    row_vals = []                                         
    for j in range(cols):                                 
        elt = gdb.parse_and_eval(f"d_A.operator()({i},{j})")
        if elt.type.code == gdb.TYPE_CODE_REF:            
            elt = elt.referenced_value()                  
        row_vals.append(str(elt))                         
    print("[" + " ".join(row_vals) + "]")                 
end
    */

/*
    python 
import gdb
m = gdb.parse_and_eval("delta")
v = gdb.parse_and_eval("grad_input")                               
rows = int(m['row'])                                      
cols = int(m['col'])                   
result = bool(True)         
for i in range(rows):                                     
    row_vals = []                                         
    row_vals2 = []
    for j in range(cols):                                 
        elt = gdb.parse_and_eval(f"delta.operator()({i},{j})")
        elt_2 = gdb.parse_and_eval(f"grad_input.operator()({i},{j})")
        if elt.type.code == gdb.TYPE_CODE_REF and elt_2.type.code == gdb.TYPE_CODE_REF:            
            elt = elt.referenced_value()                  
            elt_2 = elt_2.referenced_value()
        row_vals.append(str(elt))                                    
        row_vals2.append(str(elt_2))
    if row_vals != row_vals2:
        result = False
print(result)
end
*/

class mnist_solver_adam {
private:
    int batch_size;

    //모델 cudaStream 설정
    p2::handleStream hs;

    // --- 합성곱 층들 ---
    p2::Adam conv1_opt;      p2::ActivateLayer conv1Act;
    p2::convLayer      conv1;
    p2::Adam conv2_opt;      p2::ActivateLayer conv2Act;   
    p2::convLayer      conv2;

    // --- 완전 연결(FC) 층들 ---
    p2::Adam fc1_opt;        p2::ActivateLayer fc1Act;
    p2::PerceptronLayer fc1;       
    p2::Adam fc2_opt;        p2::ActivateLayer fc2Act;
    p2::PerceptronLayer fc2;       

    // --- 손실층 ---
    p2::LossLayer       loss;

public:
    mnist_solver_adam(int bs)
    : batch_size(bs),

      // conv1: N=bs, C=1, H=W=28, K=8, R=S=3, pad=1, stride=1
      conv1_opt(8, 1*3*3, 0.001, p2::layerType::conv),
      conv1Act(bs, 8*28*28, p2::ActType::LReLU),
      conv1(bs,1,28,28,  8,3,3, 1,1, 1,1, &conv1_opt, d2::InitType::He),

      // conv2: N=bs, C=8, H=W=28, K=16, R=S=3, pad=1, stride=2 → out:16×14×14
      conv2_opt(16, 8*3*3, 0.001, p2::layerType::conv),
      conv2Act(bs, 16*14*14, p2::ActType::LReLU),
      conv2(bs,8,28,28, 16,3,3, 1,1, 2,2, &conv2_opt, d2::InitType::He),

      // fc1: 16*14*14 → 128
      fc1_opt(16*14*14, 128, 0.001),
      fc1Act(bs, 128, p2::ActType::LReLU),
      fc1(bs, 16*14*14, 128, &fc1_opt, d2::InitType::He),

      // fc2: 128 → 10 (클래스 개수)
      fc2_opt(128, 10, 0.001),
      fc2Act(bs, 10, p2::ActType::Identity),
      fc2(bs, 128, 10, &fc2_opt, d2::InitType::He),

      // 크로스엔트로피 손실
      loss(bs, 10, p2::LossType::CrossEntropy)
    {}

    // 한 배치에 대한 순전파
    d2::d_matrix_2<double> forward(const d2::d_matrix_2<double>& X, hipStream_t str) {

        conv1.forward(X, str);
        conv1Act.pushInput(conv1.getOutput()); conv1Act.Active(str);

        // conv2
        conv2.forward(conv1Act.getOutput(), str);
        conv2Act.pushInput(conv2.getOutput()); conv2Act.Active(str);

        // 평탄화 → fc1
        auto flat = conv2Act.getOutput().reshape(batch_size, 16*14*14);
        fc1.feedforward(flat, str);
        fc1Act.pushInput(fc1.getOutput()); fc1Act.Active(str);

        // fc2
        fc2.feedforward(fc1Act.getOutput(), str);
        fc2Act.pushInput(fc2.getOutput()); fc2Act.Active(str);

        //auto forDebugfc2Act = fc2Act.getOutput();
        //forDebugfc2Act.cpyToHost();
        //auto forDebugfc2 = fc2.getOutput();
        //forDebugfc2.cpyToHost();
        //auto forDebugfc1Act = fc1Act.getOutput();
        //forDebugfc1Act.cpyToHost();
        //auto forDebugfc1 = fc1.getOutput();
        //forDebugfc1.cpyToHost();


        return fc2Act.getOutput();
    }

    // 학습 루프
    void train(d2::d_matrix_2<double>& X, d2::d_matrix_2<double>& Y, int epochs) {
        d2::d_matrix_2<double> dummy;
        int N = X.getRow();
        int B = batch_size;
        int num_batches = (N + B - 1) / B;
        auto start = std::chrono::steady_clock::now();
        std::vector<d2::d_matrix_2<double>> batch(num_batches), labels(num_batches);
        for(int i = 0; i < num_batches; ++i){
            batch[i] = X.getBatch(B, i*B);
            labels[i] = Y.getBatch(B, i*B);
            printProgressBar(i+1, num_batches, start, "batch loading... (batch " + std::to_string(i+1) + "/" + std::to_string(num_batches) + ")");
        }
        std::cout << std::endl;
        std::cout << "[batch load complete]" << std::endl;
        std::string prograss_avgloss;
        for(int e = 1; e <= epochs; ++e) {
            double avgloss = 0;
            for(int j = 0; j < num_batches; ++j){
                // 순전파
                auto Ypred = forward(batch[j], hs.model_str);

                //디버깅용
                //Ypred.cpyToHost();
                //labels[j].cpyToHost();
    
                // 손실 계산
                loss.pushOutput(Ypred);
                loss.pushTarget(labels[j]);
                double L = loss.getLoss();

                avgloss += L;
                if(std::isnan(L)){
                    std::cerr << "Loss is NaN at batch " << j << ", epoch " << e << std::endl;
                    std::cerr << "Ypred (first 10 elements): ";
                    Ypred.cpyToHost(); // Ensure host data is valid
                    for(int k=0; k<std::min(10, Ypred.size()); ++k) std::cerr << Ypred.getHostPointer()[k] << " ";
                    std::cerr << std::endl;
                    std::cerr << "labels[j] (first 10 elements): ";
                    labels[j].cpyToHost(); // Ensure host data is valid
                    for(int k=0; k<std::min(10, labels[j].size()); ++k) std::cerr << labels[j].getHostPointer()[k] << " ";
                    std::cerr << std::endl;
                    throw std::runtime_error("invalide error in loss calc.");
                }

                // 역전파: FC 층들
                auto grad2 = loss.getGrad(hs.model_str);  // dL/dz_fc2
                //grad2.cpyToHost();
                fc2.backprop(nullptr, grad2, fc2Act.d_Active(fc2.getOutput(), hs.model_str), hs.model_str);
                fc1.backprop(&fc2, dummy, fc1Act.d_Active(fc1.getOutput(), hs.model_str), hs.model_str);
    
                auto dy2 = conv2.backward(&fc1, dummy, conv2Act.d_Active(conv2.getOutput(), hs.model_str), hs.model_str);
                auto dy1 = conv1.backward(nullptr, dy2, conv1Act.d_Active(conv1.getOutput(), hs.model_str), hs.model_str);
    
                // 진행 상황 표시
                std::string prograss_batch = "batch" + std::to_string(j+1);
                std::string prograss_loss = "loss:" + std::to_string(L);
                printProgressBar(e, epochs, start, prograss_avgloss + " | " + prograss_batch + " 의 " + prograss_loss);
            }
            avgloss = avgloss/num_batches;
            prograss_avgloss = "[epoch" + std::to_string(e) + "/" + std::to_string(epochs) + "의 avgloss]:" + std::to_string(avgloss);
        }
        std::cout << std::endl;
        std::cout << "총 학습 시간: "
                  << std::chrono::duration_cast<std::chrono::seconds>(
                         std::chrono::steady_clock::now() - start
                     ).count() << "초\n";
    }
};

int main(){
    constexpr int BATCH  = 1000;
    constexpr int EPOCHS = 100;

    // MNIST 데이터 로드
    auto X = load_images_matrix("/home/sjh100/바탕화면/explab_ver2/test/train-images-idx3-ubyte");
    auto Y = load_labels_matrix("/home/sjh100/바탕화면/explab_ver2/test/train-labels-idx1-ubyte", 10);

    mnist_solver_adam solver(BATCH);
    solver.train(X, Y, EPOCHS);

    return 0;
}

